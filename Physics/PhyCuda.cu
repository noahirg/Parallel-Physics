#include "hip/hip_runtime.h"
#include "PhyCuda.cuh"


__global__
void
solveCell(CudaCircle* bodies, CudaCell* cells, const int DIV);

__device__
void
checkEleCol(unsigned id, unsigned idx, CudaCircle* bodies, CudaCell* cells);

__device__
void
solveCollision(unsigned i, unsigned j, CudaCircle* bodies);

__global__
void
updatePos(unsigned count, CudaCircle* cir, float dt);

__global__
void
applyCon(unsigned count, CudaCircle* cir, float worldSizex, float worldSizey);

__global__
void
applyForAll(unsigned count, CudaCircle* cir, float fx, float fy);


PhyCuda::PhyCuda(int sizeX, int sizeY, bool check) : PhyCWorld(sizeX, sizeY, check)
{
    /*ids = new unsigned[MAX_CIR_CU + (DIV * DIV)];
    idLoc = new unsigned[DIV * DIV]();
    hipMalloc(&ids, MAX_CIR_CU + (DIV * DIV));
    hipMalloc(&idLoc, DIV * DIV);*/
    hipMalloc(&cir, MAX_CIR_CU * sizeof(CudaCircle));
    numEle = 0;
    grid = new CudaGrid(static_cast<int>(worldSizex), static_cast<int>(worldSizey), cir, numEle);


    /**
     * need to find alternative to cudamalloc managed
     * essentially all data needs to live on the gpu during ITERC iterations in update loop
     * then data should be transfered back to cpu so it can be rendered
     * 
     */
}

PhyCuda::~PhyCuda()
{
    delete grid;
    hipFree(cir);
}

void
PhyCuda::update(float dt)
{
    const int ITERC = 8;

    //hipMemcpy(cir, &bodies[0], bodies.size() * sizeof(CudaCircle), hipMemcpyHostToDevice);

    for (int k = 0; k < ITERC; ++k)
    {
        //tempColor();
        splitCells();
        /*if (numEle > 0)
            std::cout << "piss1" << std::endl;*/
        //updateJoints(dt / static_cast<float>(ITERC));
        updatePositions(dt / static_cast<float>(ITERC));
        applyConstraint();
        grid->update(numEle);
    }

    //For render purposes
    hipMemcpy(bodies, cir, numEle * sizeof(CudaCircle), hipMemcpyDeviceToHost);
    /*if (numEle > 0)
            std::cout << "posx: " << bodies[0].posx << "    posy: " << bodies[0].posy << std::endl;*/
}

void
PhyCuda::updateJoints(float dt)
{
    for (unsigned i = 0; i < joints.size(); ++i)
    {
        joints[i].update(&cir[joints[i].cir1], &cir[joints[i].cir2], dt);
    }
}

void
PhyCuda::updatePositions(float dt)
{
    //GPUd
    int blockSize = 256;
    int numBlocks = (numEle + blockSize - 1) / blockSize;
    updatePos<<<numBlocks, blockSize>>>(numEle, cir, dt);

    hipDeviceSynchronize();
}

__global__
void
updatePos(unsigned count, CudaCircle* cir, float dt)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx >= count)
        return;

    /*if (idx == 0)
            printf("valx: %f\n", cir[0].posx);*/

    cir[idx].update(dt);
}

void
PhyCuda::applyConstraint()
{
    int blockSize = 256;
    int numBlocks = (numEle + blockSize - 1) / blockSize;
    applyCon<<<numBlocks, blockSize>>>(numEle, cir, worldSizex, worldSizey);

    hipDeviceSynchronize();
    /*for (int i = 0; i < numEle; ++i)
    {
        if (cir[i].posx > worldSizex - cir[i].rad)
            cir[i].posx = worldSizex - cir[i].rad;
        else if (cir[i].posx < cir[i].rad)
            cir[i].posx = cir[i].rad;

        if (cir[i].posy > worldSizey - cir[i].rad)
            cir[i].posy = worldSizey - cir[i].rad;
        else if (cir[i].posy < cir[i].rad)
            cir[i].posy = cir[i].rad;
    }*/
}

__global__
void
applyCon(unsigned count, CudaCircle* cir, float worldSizex, float worldSizey)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx >= count)
        return;


    if (cir[idx].posx > worldSizex - cir[idx].rad)
        cir[idx].posx = worldSizex - cir[idx].rad;
    else if (cir[idx].posx < cir[idx].rad)
        cir[idx].posx = cir[idx].rad;

    if (cir[idx].posy > worldSizey - cir[idx].rad)
        cir[idx].posy = worldSizey - cir[idx].rad;
    else if (cir[idx].posy < cir[idx].rad)
        cir[idx].posy = cir[idx].rad;
}

void
PhyCuda::applyForceAll(float fx, float fy)
{
    //GPU apply force maybe but prob not
    int blockSize = 256;
    int numBlocks = (numEle + blockSize - 1) / blockSize;
    applyForAll<<<numBlocks, blockSize>>>(numEle, cir, fx, fy);

    hipDeviceSynchronize();
    /*for (unsigned i = 0; i < numEle; ++i)
    {
        cir[i].applyForce(fx, fy);
    }*/
}

__global__
void
applyForAll(unsigned count, CudaCircle* cir, float fx, float fy)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx >= count)
        return;

    cir[idx].applyForce(fx, fy);
}

/*void
PhyCuda::tempColor()
{
    for (int i = 0; i < bodies.size(); ++i)
    {
        bodies[i].red = 255;
        bodies[i].blue = 255;
        bodies[i].green = 255;
    }
}*/

//Number of cells must be divisible by 2 * threadCount for this to work
void
PhyCuda::splitCells()
{
    //Split into 4 passes
    

        
        //loop through grid and add ids
        /*unsigned* t_ids = new unsigned[numEle + (DIV * DIV)];
        unsigned* t_idLoc = new unsigned[DIV * DIV];
        unsigned index = 0;
        //unsigned sum = 0;
        for (unsigned j = 0; j < DIV; ++j)
        {
            
            for (unsigned i = 0; i < DIV; ++i)
            {
                ids[index] = grid->m_cells[i + j * DIV].m_ids.size();

                idLoc[i + j * DIV] = index;
                //sum += index;
                unsigned kMax = ids[index];

                ++index;
                for (unsigned k = 0; k < kMax; ++k)
                {
                    ids[index] = grid->m_cells[i + j * DIV].m_ids[k];
                    //std::cout << "kag;hjsdfgfhjkl;dpsa: " << ids[index] << std::endl;
                    ++index;
                }
            }
        }

        hipMemcpy(ids, t_ids, numEle + (DIV * DIV), hipMemcpyHostToDevice);
        hipMemcpy(idLoc, t_idLoc, DIV * DIV, hipMemcpyHostToDevice);
        
        delete[] t_ids;
        delete[] t_idLoc;*/
        //Construct array of pointers to m_cells.m_ids essentially
        /*unsigned **ids = new unsigned*[cellPerPass];
        for (unsigned j = 0; j < DIV; ++j)
        {
            for (unsigned i = 0; i < DIV; ++i)
            {
                int ind = i + j * DIV + DIV + 3 + (2 * j);
                ids[i + j * DIV] = &grid->m_cells[ind].m_ids[0];

                /*ids[indexIds] = new int[grid->m_cells[ind].m_ids.size()];
                for (unsigned k = 0; k < grid->m_cells[ind].m_ids.size(); ++k)
                {
                    ids[indexIds][k] = grid->m_cells[ind].m_ids[k];
                }*-/
            }
        }*/
        //hipMallocManaged(&ids
        //hipMallocManaged(&grid, sizeof(Grid));
        
        //cir = &bodies[0];


        int cellPerPass = DIV * DIV;
        int blockSize = 256;
        int numBlocks = (cellPerPass + blockSize - 1) / blockSize;

        //solveCell<<<numBlocks, blockSize>>>(ids, idLoc, cir, DIV);
        solveCell<<<numBlocks, blockSize>>>(cir, grid->cudaCells, DIV);
        hipDeviceSynchronize();

        /*int maxInd = 0;
        for (int i = 0; i < bodies.size() + (DIV * DIV); ++i)
        {
            if (ids[i] > maxInd)
                maxInd = ids[i];
        }
        
        std::cout << "maxf: " << maxInd << std::endl;*/

        
    //}
}

__global__
void
//solveCell(unsigned* ids, unsigned* idLoc, CudaCircle* bodies, CudaCell* cells, const int DIV)
solveCell(CudaCircle* bodies, CudaCell* cells, const int DIV)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx >= DIV * DIV)
        return;


    int cellSize = cells[idx].count;

    if (cellSize == 0)
        return;


    for (int i = 0; i < cellSize; ++i)
    {
        unsigned id = cells[idx].m_ids[i];
        
        if (idx / DIV != 0 && idx % DIV != 0)
            checkEleCol(id, idx - DIV - 1, bodies, cells);
            
        if (idx / DIV != 0)
            checkEleCol(id, idx - DIV, bodies, cells);

        if (idx / DIV != 0 && (idx + 1) % DIV != 0) 
            checkEleCol(id, idx - DIV + 1, bodies, cells);

        if (idx % DIV != 0)
            checkEleCol(id, idx - 1, bodies, cells);

        checkEleCol(id, idx, bodies, cells);

        if ((idx + 1) % DIV != 0)
            checkEleCol(id, idx + 1, bodies, cells);

        if (idx / DIV != DIV - 1 && idx % DIV != 0)
            checkEleCol(id, idx + DIV - 1, bodies, cells);

        if (idx / DIV != DIV - 1)
        checkEleCol(id, idx + DIV, bodies, cells);

        if (idx / DIV != DIV - 1 && (idx + 1) % DIV != 0)
            checkEleCol(id, idx + DIV + 1, bodies, cells);
    }

    
    /*
    int nonShiftInd = threadIdx.x + blockIdx.x * blockDim.x;
    int cellSize;
    int idInd;
    if (nonShiftInd < DIV * DIV)
    {
        idInd = idLoc[nonShiftInd];
        cellSize = ids[idInd];
    }
    else
        cellSize = 0;
    //idInd where ids at the proper cell begins
    //technically its where its size is
    
    int start = idInd + 1;
    int end = idInd + cellSize + 1;
    /*max[nonShiftInd] = cellSize;
    if (nonShiftInd == 4426)
    {
        if (cellSize != 0)
        printf("start: %d,    end: %d\n", start, end);
    }*-/

    if (cellSize != 0)
    {
        for (int i = start; i < end; ++i)
        {
            unsigned id = ids[i];
            
            //grid->m_cells[ind - 1 - (DIV + 2)]
            if (nonShiftInd / DIV != 0 && nonShiftInd % DIV != 0)
                checkEleCol(id, ids, idLoc[nonShiftInd - DIV - 1], bodies);
                
            if (nonShiftInd / DIV != 0)
                checkEleCol(id, ids, idLoc[nonShiftInd - DIV], bodies);

            if (nonShiftInd / DIV != 0 && (nonShiftInd + 1) % DIV != 0) 
                checkEleCol(id, ids, idLoc[nonShiftInd - DIV + 1], bodies);

            if (nonShiftInd % DIV != 0)
                checkEleCol(id, ids, idLoc[nonShiftInd - 1], bodies);

            checkEleCol(id, ids, idLoc[nonShiftInd], bodies);

            if ((nonShiftInd + 1) % DIV != 0)
                checkEleCol(id, ids, idLoc[nonShiftInd + 1], bodies);

            if (nonShiftInd / DIV != DIV - 1 && nonShiftInd % DIV != 0)
                checkEleCol(id, ids, idLoc[nonShiftInd + DIV - 1], bodies);

            if (nonShiftInd / DIV != DIV - 1)
            checkEleCol(id, ids, idLoc[nonShiftInd + DIV], bodies);

            if (nonShiftInd / DIV != DIV - 1 && (nonShiftInd + 1) % DIV != 0)
                checkEleCol(id, ids, idLoc[nonShiftInd + DIV + 1], bodies);
        }
    }*/
}

/*__global__
void
solveCell(Grid* grid, Circle* bodies, const int DIV)
{
    int ind = threadIdx.x + blockIdx.x * DIV + DIV + 3 + (2 * blockIdx.x);
    if (grid->m_cells[ind].m_ids.size() != 0)
    {
        for (unsigned i = 0; i < grid->m_cells[ind].m_ids.size(); ++i)
        {
            unsigned id = grid->m_cells[ind].m_ids[i];
            checkEleCol(id, grid->m_cells[ind - 1 - (DIV + 2)], bodies);
            checkEleCol(id, grid->m_cells[ind     - (DIV + 2)], bodies);
            checkEleCol(id, grid->m_cells[ind + 1 - (DIV + 2)], bodies);
            checkEleCol(id, grid->m_cells[ind             - 1], bodies);
            checkEleCol(id, grid->m_cells[ind                ], bodies);
            checkEleCol(id, grid->m_cells[ind             + 1], bodies);
            checkEleCol(id, grid->m_cells[ind - 1 + (DIV + 2)], bodies);
            checkEleCol(id, grid->m_cells[ind     + (DIV + 2)], bodies);
            checkEleCol(id, grid->m_cells[ind + 1 + (DIV + 2)], bodies);
        }
    }
}*/

__device__
void
checkEleCol(unsigned id, unsigned idx, CudaCircle* bodies, CudaCell* cells)
{
    int cellSize = cells[idx].count;
    
    for (int i = 0; i < cellSize; ++i)
    {
        solveCollision(id, cells[idx].m_ids[i], bodies);
    }
}

/*__device__
void
checkEleCol(unsigned id, unsigned* ids, unsigned start, CudaCircle* bodies)
{
    int cellSize = ids[start];
    int begin = start + 1;
    int end = start + cellSize + 1;
    
        

    for (int i = begin; i < end; ++i)
    {
        solveCollision(id, ids[i], bodies);
    }
    //printf("id: %d\n", id);
}*/

__device__
void
solveCollision(unsigned i, unsigned j, CudaCircle* bodies)
{
    float epsilon = .0001f;

    /*if (i == 299)
    {
        bodies[j].red = 0;
    }*/
    if (i == j)
        return;

    
    //printf("i: %d     bodyx: %f       bodyy: %f\n", i, bodies[i].posx, bodies[i].posy);
    
    float colAxisx = bodies[i].posx - bodies[j].posx;
    float colAxisy = bodies[i].posy - bodies[j].posy;
    float distSq = colAxisx * colAxisx + colAxisy * colAxisy;
    //for poly - get line it crossed 
    // push the shape along the normal of that line
    float iRad = bodies[i].rad;
    float jRad = bodies[j].rad;
    //float jRad = bodies[i].getRad(bodies[j].pos);
    //float iRad = bodies[j].getRad(bodies[i].pos);
    float radD = iRad + jRad;
    if (distSq < radD * radD && distSq > epsilon)
    {
        float dist = sqrtf(distSq);
        float normalx = colAxisx / dist;
        float normaly = colAxisy / dist;
        float delta = radD - dist;
        float di = (jRad / radD) * delta;
        float dj = (iRad / radD) * delta;

        if (bodies[i].pinned && bodies[j].pinned)
            {di = 0; dj = 0;}
        else if (bodies[i].pinned)
            dj = delta;
        else if (bodies[j].pinned)
            di = delta;

        
        bodies[i].posx += di * normalx;
        bodies[i].posy += di * normaly;
        bodies[j].posx -= dj * normalx;
        bodies[j].posy -= dj * normaly;
    }
}

CudaCircle* 
PhyCuda::createCircle(float posx, float posy, float mass, float rad, bool pinned)
{
    //bodies.emplace_back( posx, posy, mass, rad, pinned );
    //grid->addSingle(posx, posy, bodies.size() - 1);

    CudaCircle circ (posx, posy, mass, rad, pinned);
    hipMemcpy(&(cir[numEle]), &circ, sizeof(CudaCircle), hipMemcpyHostToDevice);
    ++numEle;

    //grid->update(numEle);
    //grid->addSingle(posx, posy, numEle - 1);

    return {};//&cir[numEle - 1];
}

void
PhyCuda::insertToGrid(float posx, float posy, unsigned id)
{
    grid->addSingle(posx, posy, id);
}

/*std::vector<std::array<int, 4>>
PhyCuda::getGrid()
{
    return grid->getCells();
}
*/